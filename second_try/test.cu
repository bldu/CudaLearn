#include <iostream>
#include <hip/hip_runtime.h>

// 矩阵乘法的CUDA核函数
__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    const int N = 1024;  // 矩阵的大小
    size_t size = N * N * sizeof(float);

    // 分配主机内存
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // 初始化矩阵A和B
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 将数据从主机传输到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义CUDA的线程块和网格大小
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 调用CUDA核函数
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 将结果从设备传输回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 打印结果（仅用于验证）
    for (int i = 0; i < 10; ++i) {
        for (int j = 0; j < 10; ++j) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // 释放内存
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
